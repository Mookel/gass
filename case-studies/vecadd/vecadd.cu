
#include <hip/hip_runtime.h>
__global__ void vecadd(float *C, float *A, float *B, int N) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  if (tid < N)
    C[tid] = A[tid] * B[tid];
}